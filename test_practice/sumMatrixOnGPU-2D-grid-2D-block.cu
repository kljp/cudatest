#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_prac.h"

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny){

	float *ia = A;
	float *ib = B;
	float *ic = C;

	for(int iy = 0; iy < ny; iy++)
	{
		for(int ix = 0; ix < nx; ix++)
				ic[ix] = ia[ix] + ib[ix];

		ia += nx;
		ib += nx;
		ic += nx;
	}
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny){

	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * nx + ix;

	if(ix < nx && iy < ny)
		MatC[idx] = MatA[idx] + MatB[idx];
}

int main(int argc, char** argv){

	printf("%s Starting...\n", argv[0]);

	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));
	
	int nx = 2048;
	int ny = 2048;

	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);
	printf("Matrix size: nx %d ny %d\n", nx, ny);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A =			(float *) malloc(nBytes);
	h_B =			(float *) malloc(nBytes);
	hostRef =	(float *) malloc(nBytes);
	gpuRef =		(float *) malloc(nBytes);

	double iStart = cpuSecond();
	initialData(h_A, nxy);
	initialData(h_B, nxy);
	double iElaps = cpuSecond() - iStart;

	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	iStart = cpuSecond();
	sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
	iElaps = cpuSecond() - iStart;
	printf("sumMatrixOnHost elapsed %f sec\n", iElaps);

	float *d_MatA, *d_MatB, *d_MatC;
	hipMalloc((void **) &d_MatA, nBytes);
	hipMalloc((void **) &d_MatB, nBytes);
	hipMalloc((void **) &d_MatC, nBytes);

	hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

	int dimx = 32;
	int dimy = 16;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) /block.x, (ny + block.y - 1) / block.y);

	iStart = cpuSecond();
	sumMatrixOnGPU2D <<<grid, block>>> (d_MatA, d_MatB, d_MatC, nx, ny);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("sumMatrixOnGPU2D <<< (%d, %d), (%d, %d)>>> elapsed %f sec\n", grid.x, grid.y, block.x, block.y, iElaps);

	hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

	checkResult(hostRef, gpuRef, nxy);

	hipFree(d_MatA);
	hipFree(d_MatB);
	hipFree(d_MatC);

	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	hipDeviceReset();

	return 0;
}
