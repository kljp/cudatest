#define N 16


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

using namespace std;

__global__ void add(int *a, int *b, int *c){

	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
	printf("%d		blockIdx=%d\n", c[blockIdx.x], blockIdx.x);
}

__global__ void build_array(int **a, int **b, int **c){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	c[i][j] = 3;
	printf("%d		index: %2d, %2d		block: %2d, %2d		thread: %2d, %2d\n", c[i][j], i, j, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

void random_ints(int *array, int size){

	int i;
	for(i = 0; i < size; i++)
		array[i] = rand() % 10;
}

int main()
{
	int **a, **b, **c;
	int **gpu_a, **gpu_b, **gpu_c;
	
	a = (int **) malloc(sizeof(int *) * N);
	b = (int **) malloc(sizeof(int *) * N);
	c = (int **) malloc(sizeof(int *) * N);

	hipMalloc((void **) &gpu_a, sizeof(int *) * N);
	hipMalloc((void **) &gpu_b, sizeof(int *) * N);
	hipMalloc((void **) &gpu_c, sizeof(int *) * N);

	for(int i = 0; i < N; i++)
	{
		//a[i] = (int *) malloc(sizeof(int) * N);
		//b[i] = (int *) malloc(sizeof(int) * N);
		//c[i] = (int *) malloc(sizeof(int) * N);

		hipMalloc((void **) &a[i], sizeof(int) * N);
		hipMalloc((void **) &b[i], sizeof(int) * N);
		hipMalloc((void **) &c[i], sizeof(int) * N);
	}

	hipMemcpy(gpu_b, b, sizeof(int *) * N, hipMemcpyHostToDevice);
	hipMemcpy(gpu_c, c, sizeof(int *) * N, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
	build_array<<<numBlocks, threadsPerBlock>>>(gpu_a, gpu_b, gpu_c);

	hipMemcpy(c, gpu_c, sizeof(int *) * N, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
			cout << c[i][j] << endl;//printf("%d		index: %d, %d\n", c[i][j], i, j);
	}
	
	for(int i = 0; i < N; i++)
	{
		hipFree(a[i]);
		hipFree(b[i]);
		hipFree(c[i]);
	}

	free(a);
	free(b);
	free(c);

	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);

	return 0;
}
